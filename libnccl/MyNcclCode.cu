#include "MyNcclCode.h"


extern "C" {

void MyncclSend(const void* sendbuff, size_t count, MyNcclDataType_t datatype, int peer, MyNcclComm_t mycomm, MycudaStream_t stream) {
                   ncclSend(sendbuff, count, datatype, peer, mycomm.ncclComm, stream);
}


void MyncclRecv(void* recvbuff, size_t count, MyNcclDataType_t datatype, int peer, MyNcclComm_t mycomm, MycudaStream_t stream) {
                   ncclRecv(recvbuff, count, datatype, peer, mycomm.ncclComm, stream);
}


void MyncclGetUniqueId(MyncclUniqueId Myout) {
                  ncclGetUniqueId(&(Myout.ncclId)); 
}


void MyncclCommInitRank(MyNcclComm_t mycomm, int nranks, MyncclUniqueId Myout, int myrank) {
                  ncclCommInitRank(&(mycomm.ncclComm), nranks, Myout.ncclId, myrank);
}

void MycudaStreamCreate ( MycudaStream_t* MypStream ) {
                  hipStreamCreate (MypStream); 
}


void MycudaStreamDestroy ( MycudaStream_t MypStream_create ) {
                  hipStreamDestroy (MypStream_create); 
}


void MyncclCommDestroy (MyNcclComm_t mycomm) {
                  ncclCommDestroy(mycomm.ncclComm);
}

void MycudaMalloc ( void** devPtr, size_t size ) {
                 hipMalloc (devPtr, size); 
} 

void MycudaFree ( void* devPtr_free ) {
                  hipFree (devPtr_free);
}

void MycudaStreamSynchronize ( MycudaStream_t Mystream ) {
                  hipStreamSynchronize (Mystream);
}

void MycudaMemcpy ( void* dst, const void* src, size_t count, MycudaMemcpyKind kind ) {
                  hipMemcpyKind cuda_kind;
                  if (kind == MycudaMemcpyHostToHost){
                                    cuda_kind = hipMemcpyHostToHost;
                  } else if (kind == MycudaMemcpyHostToDevice) {
                                    cuda_kind = hipMemcpyHostToDevice;
                  } else if (kind == MycudaMemcpyDeviceToHost) {
                                    cuda_kind = hipMemcpyDeviceToHost;
                  } else if (kind == MycudaMemcpyDeviceToDevice) {
                                    cuda_kind = hipMemcpyDeviceToDevice;
                  } else if (kind == MycudaMemcpyDefault) {
                                    cuda_kind = hipMemcpyDefault;
                  }

                //   hipMemcpyKind cuda_kind = static_cast<hipMemcpyKind>(kind);

                  hipMemcpy (dst, src, count, cuda_kind);
}

}


