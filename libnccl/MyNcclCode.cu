#include "MyNcclCode.h"


extern "C" {

ncclResult_t MyncclSend(const void* sendbuff, size_t count, MyNcclDataType_t datatype, int peer, MyNcclComm_t mycomm, MycudaStream_t stream) {
                   return ncclSend(sendbuff, count, datatype, peer, mycomm.ncclComm, stream);
}


ncclResult_t MyncclRecv(void* recvbuff, size_t count, MyNcclDataType_t datatype, int peer, MyNcclComm_t mycomm, MycudaStream_t stream) {
                   return ncclRecv(recvbuff, count, datatype, peer, mycomm.ncclComm, stream);
}


ncclResult_t MyncclGetUniqueId(MyncclUniqueId* Myout) {
                  return ncclGetUniqueId(&(Myout->ncclId)); 
}


ncclResult_t MyncclCommInitRank(MyNcclComm_t* mycomm_t, int nranks, MyncclUniqueId Myout, int myrank) {
                  return ncclCommInitRank(&(mycomm_t->ncclComm), nranks, Myout.ncclId, myrank);
}

hipError_t MycudaStreamCreate ( MycudaStream_t* MypStream ) {
                  return hipStreamCreate(MypStream); 
}


hipError_t MycudaStreamDestroy ( MycudaStream_t MypStream_create ) {
                  return hipStreamDestroy (MypStream_create); 
}


ncclResult_t MyncclCommDestroy (MyNcclComm_t mycomm) {
                  return ncclCommDestroy(mycomm.ncclComm);
}

hipError_t MycudaMalloc ( void** devPtr, size_t size ) {
                 return hipMalloc (devPtr, size); 
} 

hipError_t MycudaFree ( void* devPtr_free ) {
                  return hipFree (devPtr_free);
}

hipError_t MycudaStreamSynchronize ( MycudaStream_t Mystream ) {
                  return hipStreamSynchronize (Mystream);
}

hipError_t MycudaMemcpy ( void* dst, const void* src, size_t count, MycudaMemcpyKind kind ) {
                  hipMemcpyKind cuda_kind;
                  if (kind == MycudaMemcpyHostToHost){
                                    cuda_kind = hipMemcpyHostToHost;
                  } else if (kind == MycudaMemcpyHostToDevice) {
                                    cuda_kind = hipMemcpyHostToDevice;
                  } else if (kind == MycudaMemcpyDeviceToHost) {
                                    cuda_kind = hipMemcpyDeviceToHost;
                  } else if (kind == MycudaMemcpyDeviceToDevice) {
                                    cuda_kind = hipMemcpyDeviceToDevice;
                  } else if (kind == MycudaMemcpyDefault) {
                                    cuda_kind = hipMemcpyDefault;
                  }

                //   hipMemcpyKind cuda_kind = static_cast<hipMemcpyKind>(kind);

                  return hipMemcpy (dst, src, count, cuda_kind);
}

uint64_t NCCL_getHostHash(const char* hostname) {
    uint64_t hash = 5381;
    int c;

    while ((c = *hostname++)) {
        hash = ((hash << 5) + hash) + c; /* hash * 33 + c */
    }

    return hash;
}

}


